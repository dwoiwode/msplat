#include "hip/hip_runtime.h"
#include "rasterizer_impl.h"
#include <iostream>
#include <fstream>
#include <algorithm>
#include <numeric>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <hipcub/hipcub.hpp>
#include <cub/device/device_radix_sort.cuh>
#define GLM_FORCE_CUDA
#include <glm/glm.hpp>

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;
#include "config.h"
#include "auxiliary.h"


// Helper function to find the next-highest bit of the MSB
// on the CPU.
uint32_t getHigherMsb(uint32_t n)
{
	uint32_t msb = sizeof(n) * 4;
	uint32_t step = msb;
	while (step > 1)
	{
		step /= 2;
		if (n >> msb)
			msb += step;
		else
			msb -= step;
	}
	if (n >> msb)
		msb++;
	return msb;
}


// Generates one key/value pair for all Gaussian / tile overlaps. 
// Run once per Gaussian (1:N mapping).
__global__ void duplicateWithKeys(
	const int P,
	const float2* points_xy,
	const float* depths,
	const uint32_t* offsets,
	uint64_t* gaussian_keys_unsorted,
	uint32_t* gaussian_values_unsorted,
	const int* radii,
	const bool* visibility_filter,
	dim3 grid)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P || !visibility_filter[idx])
		return;

	// Find this Gaussian's offset in buffer for writing keys/values.
	uint32_t off = (idx == 0) ? 0 : offsets[idx - 1];
	uint2 rect_min, rect_max;

	getRect(points_xy[idx], radii[idx], rect_min, rect_max, grid);

	// For each tile that the bounding rect overlaps, emit a 
	// key/value pair. The key is |  tile ID  |      depth      |,
	// and the value is the ID of the Gaussian. Sorting the values 
	// with this key yields Gaussian IDs in a list, such that they
	// are first sorted by tile and then by depth. 
	for (int y = rect_min.y; y < rect_max.y; y++)
	{
		for (int x = rect_min.x; x < rect_max.x; x++)
		{
			uint64_t key = y * grid.x + x;
			key <<= 32;
			key |= *((uint32_t*)&depths[idx]);
			gaussian_keys_unsorted[off] = key;
			gaussian_values_unsorted[off] = idx;
			off++;
		}
	}
}

// Check keys to see if it is at the start/end of one tile's range in 
// the full sorted list. If yes, write start/end of this tile. 
// Run once per instanced (duplicated) Gaussian ID.
__global__ void identifyTileRanges(int L, uint64_t* point_list_keys, uint2* ranges)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= L)
		return;

	// Read tile ID from key. Update start/end of tile range if at limit.
	uint64_t key = point_list_keys[idx];
	uint32_t currtile = key >> 32;
	if (idx == 0)
		ranges[currtile].x = 0;
	else
	{
		uint32_t prevtile = point_list_keys[idx - 1] >> 32;
		if (currtile != prevtile)
		{
			ranges[prevtile].y = idx;
			ranges[currtile].x = idx;
		}
	}
	if (idx == L - 1)
		ranges[currtile].y = L;
}


template <uint32_t CHANNELS>
__global__ void __launch_bounds__(BLOCK_X * BLOCK_Y)
RenderForwardCUDAKernel(
	const uint2* __restrict__ ranges,
	const uint32_t* __restrict__ point_list,
	int W, int H,
	const float2* __restrict__ means2D,
	const float* __restrict__ features,
	const float3* __restrict__ conics,
	const float* __restrict__ opacities,
	float* __restrict__ final_T,
	uint32_t* __restrict__ n_contrib,
	float* __restrict__ out_feature)
{
	// Identify current tile and associated min/max pixel range.
	auto block = cg::this_thread_block();
	uint32_t horizontal_blocks = (W + BLOCK_X - 1) / BLOCK_X;
	uint2 pix_min = { block.group_index().x * BLOCK_X, block.group_index().y * BLOCK_Y };
	uint2 pix_max = { min(pix_min.x + BLOCK_X, W), min(pix_min.y + BLOCK_Y , H) };
	uint2 pix = { pix_min.x + block.thread_index().x, pix_min.y + block.thread_index().y };
	uint32_t pix_id = W * pix.y + pix.x;
	float2 pixf = { (float)pix.x, (float)pix.y };

	// Check if this thread is associated with a valid pixel or outside.
	bool inside = pix.x < W&& pix.y < H;
	// Done threads can help with fetching, but don't rasterize
	bool done = !inside;

	// Load start/end range of IDs to process in bit sorted list.
	uint2 range = ranges[block.group_index().y * horizontal_blocks + block.group_index().x];
	const int rounds = ((range.y - range.x + BLOCK_SIZE - 1) / BLOCK_SIZE);
	int toDo = range.y - range.x;

	// Allocate storage for batches of collectively fetched data.
	__shared__ int collected_id[BLOCK_SIZE];
	__shared__ float2 collected_xy[BLOCK_SIZE];
	__shared__ float3 collected_conics[BLOCK_SIZE];
	__shared__ float collected_opacities[BLOCK_SIZE];

	// Initialize helper variables
	float T = 1.0f;
	uint32_t contributor = 0;
	uint32_t last_contributor = 0;
	float F[CHANNELS] = { 0 };

	// Iterate over batches until all done or range is complete
	for (int i = 0; i < rounds; i++, toDo -= BLOCK_SIZE)
	{
		// End if entire block votes that it is done rasterizing
		int num_done = __syncthreads_count(done);
		if (num_done == BLOCK_SIZE)
			break;

		// Collectively fetch per-Gaussian data from global to shared
		int progress = i * BLOCK_SIZE + block.thread_rank();
		if (range.x + progress < range.y)
		{
			int coll_id = point_list[range.x + progress];
			collected_id[block.thread_rank()] = coll_id;
			collected_xy[block.thread_rank()] = means2D[coll_id];
			collected_conics[block.thread_rank()] = conics[coll_id];
			collected_opacities[block.thread_rank()] = opacities[coll_id];
		}
		block.sync();

		// Iterate over current batch
		for (int j = 0; !done && j < min(BLOCK_SIZE, toDo); j++)
		{
			// Keep track of current position in range
			contributor++;

			// Resample using conic matrix (cf. "Surface 
			// Splatting" by Zwicker et al., 2001)
			float2 xy = collected_xy[j];
			float2 d = { xy.x - pixf.x, xy.y - pixf.y };
			float3 conic = collected_conics[j];
			float opacity = collected_opacities[j];
			float power = -0.5f * (conic.x * d.x * d.x + conic.z * d.y * d.y) - conic.y * d.x * d.y;
			if (power > 0.0f)
				continue;

			// Eq. (2) from 3D Gaussian splatting paper.
			// Obtain alpha by multiplying with Gaussian opacity
			// and its exponential falloff from mean.
			// Avoid numerical instabilities (see paper appendix). 
			float alpha = min(0.99f, opacity * exp(power));
			if (alpha < 1.0f / 255.0f)
				continue;
			float test_T = T * (1 - alpha);
			if (test_T < 0.0001f)
			{
				done = true;
				continue;
			}

			// Eq. (3) from 3D Gaussian splatting paper.
			for (int ch = 0; ch < CHANNELS; ch++)
				F[ch] += features[collected_id[j] * CHANNELS + ch] * alpha * T;

			T = test_T;

			// Keep track of last range entry to update this
			// pixel.
			last_contributor = contributor;
		}
	}

	// All threads that treat valid pixel write out their final
	// rendering data to the frame and auxiliary buffers.
	if (inside)
	{
		final_T[pix_id] = T;
		n_contrib[pix_id] = last_contributor;
		for (int ch = 0; ch < CHANNELS; ch++)
			out_feature[ch * H * W + pix_id] = F[ch];
	}
}

template <uint32_t C>
__global__ void __launch_bounds__(BLOCK_X * BLOCK_Y)
RenderBackwardCUDAKernel(
	const uint2* __restrict__ ranges,
	const uint32_t* __restrict__ point_list,
	int W, int H,
	const float2* __restrict__ means2D,
	const float3* __restrict__ conics,
	const float* __restrict__ opacities,
	const float* __restrict__ features,
	const float* __restrict__ final_Ts,
	const uint32_t* __restrict__ n_contrib,
	const float* __restrict__ dL_dout_feature,
	float* __restrict__ dL_dfeatures,
	float2* __restrict__ dL_dmeans2D,
	float3* __restrict__ dL_dconics,
	float* __restrict__ dL_dopacities
	)
{
	// We rasterize again. Compute necessary block info.
	auto block = cg::this_thread_block();
	const uint32_t horizontal_blocks = (W + BLOCK_X - 1) / BLOCK_X;
	const uint2 pix_min = { block.group_index().x * BLOCK_X, block.group_index().y * BLOCK_Y };
	const uint2 pix_max = { min(pix_min.x + BLOCK_X, W), min(pix_min.y + BLOCK_Y , H) };
	const uint2 pix = { pix_min.x + block.thread_index().x, pix_min.y + block.thread_index().y };
	const uint32_t pix_id = W * pix.y + pix.x;
	const float2 pixf = { (float)pix.x, (float)pix.y };

	const bool inside = pix.x < W&& pix.y < H;
	const uint2 range = ranges[block.group_index().y * horizontal_blocks + block.group_index().x];

	const int rounds = ((range.y - range.x + BLOCK_SIZE - 1) / BLOCK_SIZE);

	bool done = !inside;
	int toDo = range.y - range.x;

	__shared__ int collected_id[BLOCK_SIZE];
	__shared__ float2 collected_xy[BLOCK_SIZE];
	__shared__ float3 collected_conics[BLOCK_SIZE];
	__shared__ float collected_opacities[BLOCK_SIZE];
	__shared__ float collected_features[C * BLOCK_SIZE];

	// In the forward, we stored the final value for T, the
	// product of all (1 - alpha) factors. 
	const float T_final = inside ? final_Ts[pix_id] : 0;
	float T = T_final;

	// We start from the back. The ID of the last contributing
	// Gaussian is known from each pixel from the forward.
	uint32_t contributor = toDo;
	const int last_contributor = inside ? n_contrib[pix_id] : 0;

	float accum_rec[C] = { 0 };
	float dL_dpixel[C];
	if (inside)
		for (int i = 0; i < C; i++)
			dL_dpixel[i] = dL_dout_feature[i * H * W + pix_id];

	float last_alpha = 0;
	float last_color[C] = { 0 };

	// Gradient of pixel coordinate w.r.t. normalized 
	// screen-space viewport corrdinates (-1 to 1)
	const float ddelx_dx = 0.5 * W;
	const float ddely_dy = 0.5 * H;

	// Traverse all Gaussians
	for (int i = 0; i < rounds; i++, toDo -= BLOCK_SIZE)
	{
		// Load auxiliary data into shared memory, start in the BACK
		// and load them in revers order.
		block.sync();
		const int progress = i * BLOCK_SIZE + block.thread_rank();
		if (range.x + progress < range.y)
		{
			const int coll_id = point_list[range.y - progress - 1];
			collected_id[block.thread_rank()] = coll_id;
			collected_xy[block.thread_rank()] = means2D[coll_id];
			collected_conics[block.thread_rank()] = conics[coll_id];
			collected_opacities[block.thread_rank()] = opacities[coll_id];
			for (int i = 0; i < C; i++)
				collected_features[i * BLOCK_SIZE + block.thread_rank()] = features[coll_id * C + i];
		}
		block.sync();

		// Iterate over Gaussians
		for (int j = 0; !done && j < min(BLOCK_SIZE, toDo); j++)
		{
			// Keep track of current Gaussian ID. Skip, if this one
			// is behind the last contributor for this pixel.
			contributor--;
			if (contributor >= last_contributor)
				continue;

			// Compute blending values, as before.
			const float2 xy = collected_xy[j];
			const float2 d = { xy.x - pixf.x, xy.y - pixf.y };
			const float3 conic = collected_conics[j];
			const float opacity = collected_opacities[j];
			const float power = -0.5f * (conic.x * d.x * d.x + conic.z * d.y * d.y) - conic.y * d.x * d.y;
			if (power > 0.0f)
				continue;

			const float G = exp(power);
			const float alpha = min(0.99f, opacity * G);
			if (alpha < 1.0f / 255.0f)
				continue;

			T = T / (1.f - alpha);
			const float dchannel_dcolor = alpha * T;

			// Propagate gradients to per-Gaussian features and keep
			// gradients w.r.t. alpha (blending factor for a Gaussian/pixel
			// pair).
			float dL_dalpha = 0.0f;
			const int global_id = collected_id[j];
			for (int ch = 0; ch < C; ch++)
			{
				const float c = collected_features[ch * BLOCK_SIZE + j];
				// Update last color (to be used in the next iteration)
				accum_rec[ch] = last_alpha * last_color[ch] + (1.f - last_alpha) * accum_rec[ch];
				last_color[ch] = c;

				const float dL_dchannel = dL_dpixel[ch];
				dL_dalpha += (c - accum_rec[ch]) * dL_dchannel;
				// Update the gradients w.r.t. color of the Gaussian. 
				// Atomic, since this pixel is just one of potentially
				// many that were affected by this Gaussian.
				atomicAdd(&(dL_dfeatures[global_id * C + ch]), dchannel_dcolor * dL_dchannel);
			}
			dL_dalpha *= T;
			// Update last alpha (to be used in the next iteration)
			last_alpha = alpha;

			// Helpful reusable temporary variables
			const float dL_dG = opacity * dL_dalpha;
			const float gdx = G * d.x;
			const float gdy = G * d.y;
			const float dG_ddelx = -gdx * conic.x - gdy * conic.y;
			const float dG_ddely = -gdy * conic.z - gdx * conic.y;

			// Update gradients w.r.t. 2D mean position of the Gaussian
			atomicAdd(&dL_dmeans2D[global_id].x, dL_dG * dG_ddelx * ddelx_dx);
			atomicAdd(&dL_dmeans2D[global_id].y, dL_dG * dG_ddely * ddely_dy);

			// Update gradients w.r.t. 2D covariance (2x2 matrix, symmetric)
			atomicAdd(&dL_dconics[global_id].x, -0.5f * gdx * d.x * dL_dG);
			atomicAdd(&dL_dconics[global_id].y, -0.5f * gdx * d.y * dL_dG);
			atomicAdd(&dL_dconics[global_id].z, -0.5f * gdy * d.y * dL_dG);

			// Update gradients w.r.t. opacity of the Gaussian
			atomicAdd(&(dL_dopacities[global_id]), G * dL_dalpha);
		}
	}
}

int RenderForwardCUDA(
	const int P,
	const int width, const int height,
	const float* features,
	const float* depths,
	const int32_t* radii,
	const float2* means2D,
	const float3* conics,
	const float* opacities,
	const int32_t* tiles_touched,
	const bool* visibility_filter,
	std::function<char* (size_t)> binningBuffer,
	std::function<char* (size_t)> imageBuffer,
	float* out_feature,
	bool debug)
{
	dim3 tile_grid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y, 1);
	dim3 block(BLOCK_X, BLOCK_Y, 1);

	// Dynamically resize image-based auxiliary buffers during training
	size_t img_chunk_size = required<ImageState>(width * height);
	char* img_chunkptr = imageBuffer(img_chunk_size);
	ImageState imgState = ImageState::fromChunk(img_chunkptr, width * height);

	char* scanning_space;
	uint32_t* point_offsets;
    hipMalloc((void**)&scanning_space, P * sizeof(char));
    hipMalloc((void**)&point_offsets, P * sizeof(uint32_t));

	size_t temp_storage_bytes = 0;
	hipcub::DeviceScan::InclusiveSum(nullptr, temp_storage_bytes, point_offsets, point_offsets, P);

	// Compute prefix sum over full list of touched tile counts by Gaussians
	// E.g., [2, 3, 0, 2, 1] -> [2, 5, 5, 7, 8]
	CHECK_CUDA(hipcub::DeviceScan::InclusiveSum(scanning_space, temp_storage_bytes, tiles_touched, point_offsets, P), debug)

	// Retrieve total number of Gaussian instances to launch and resize aux buffers
	int num_rendered;
	CHECK_CUDA(hipMemcpy(&num_rendered, point_offsets + P - 1, sizeof(int), hipMemcpyDeviceToHost), debug);

	size_t binning_chunk_size = required<BinningState>(num_rendered);
	char* binning_chunkptr = binningBuffer(binning_chunk_size);
	BinningState binningState = BinningState::fromChunk(binning_chunkptr, num_rendered);

	// For each instance to be rendered, produce adequate [ tile | depth ] key 
	// and corresponding dublicated Gaussian indices to be sorted
	duplicateWithKeys << <(P + 255) / 256, 256 >> > (
		P,
		means2D,
		depths,
		point_offsets,
		binningState.point_list_keys_unsorted,
		binningState.point_list_unsorted,
		radii,
		visibility_filter,
		tile_grid)
	CHECK_CUDA(, debug)

	int bit = getHigherMsb(tile_grid.x * tile_grid.y);

	// Sort complete list of (duplicated) Gaussian indices by keys
	CHECK_CUDA(hipcub::DeviceRadixSort::SortPairs(
		binningState.list_sorting_space,
		binningState.sorting_size,
		binningState.point_list_keys_unsorted, binningState.point_list_keys,
		binningState.point_list_unsorted, binningState.point_list,
		num_rendered, 0, 32 + bit), debug)

	CHECK_CUDA(hipMemset(imgState.ranges, 0, tile_grid.x * tile_grid.y * sizeof(uint2)), debug);

	// Identify start and end of per-tile workloads in sorted list
	if (num_rendered > 0)
		identifyTileRanges << <(num_rendered + 255) / 256, 256 >> > (
			num_rendered,
			binningState.point_list_keys,
			imgState.ranges);
	CHECK_CUDA(, debug)

    RenderForwardCUDAKernel<NUM_CHANNELS> <<<tile_grid, block >> > (
		imgState.ranges,
		binningState.point_list,
		width, height,
		means2D,
		features,
		conics,
		opacities,
		imgState.accum_alpha,
		imgState.n_contrib,
		out_feature);
	
	hipFree(scanning_space);
	hipFree(point_offsets);

	return num_rendered;
}


void RenderBackwardCUDA(
	const int P, const int num_rendered,
	const int width, const int height,
	const float* features,
	const float2* means2D,
	const float3* conics,
	const float* opacities,
	char* binning_buffer,
	char* img_buffer,
	const float* dL_dout_feature,
	float* dL_dfeatures,
	float2* dL_dmeans2D,
	float3* dL_dconics,
	float* dL_dopacities,
	bool debug)
{
	BinningState binningState = BinningState::fromChunk(binning_buffer, num_rendered);
	ImageState imgState = ImageState::fromChunk(img_buffer, width * height);

	const dim3 tile_grid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y, 1);
	const dim3 block(BLOCK_X, BLOCK_Y, 1);

	RenderBackwardCUDAKernel<NUM_CHANNELS> <<<tile_grid, block>>>(
		imgState.ranges,
		binningState.point_list,
		width, height,
		means2D,
		conics,
		opacities,
		features,
		imgState.accum_alpha,
		imgState.n_contrib,
		dL_dout_feature,
		dL_dfeatures,
		dL_dmeans2D,
		dL_dconics,
		dL_dopacities
	);
}