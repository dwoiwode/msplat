#include "hip/hip_runtime.h"
#include "rasterizer_impl.h"
#include <iostream>
#include <fstream>
#include <algorithm>
#include <numeric>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <hipcub/hipcub.hpp>
#include <cub/device/device_radix_sort.cuh>
#define GLM_FORCE_CUDA
#include <glm/glm.hpp>

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;
#include "config.h"
#include "auxiliary.h"


ImageState ImageState::fromChunk(char*& chunk, size_t N)
{
	ImageState img;
	obtain(chunk, img.accum_alpha, N, 128);
	obtain(chunk, img.n_contrib, N, 128);
	obtain(chunk, img.ranges, N, 128);
	return img;
}

BinningState BinningState::fromChunk(char*& chunk, size_t P)
{
	BinningState binning;
	obtain(chunk, binning.point_list, P, 128);
	obtain(chunk, binning.point_list_unsorted, P, 128);
	obtain(chunk, binning.point_list_keys, P, 128);
	obtain(chunk, binning.point_list_keys_unsorted, P, 128);
	hipcub::DeviceRadixSort::SortPairs(
		nullptr, binning.sorting_size,
		binning.point_list_keys_unsorted, binning.point_list_keys,
		binning.point_list_unsorted, binning.point_list, P);
	obtain(chunk, binning.list_sorting_space, binning.sorting_size, 128);
	return binning;
}

__forceinline__ __device__ bool in_frustum(int idx,
	const float* orig_points,
	const float* viewmatrix,
	const float* projmatrix,
	float3& p_view)
{
	float3 p_orig = { orig_points[3 * idx], orig_points[3 * idx + 1], orig_points[3 * idx + 2] };

	// Bring points to screen space
	float4 p_hom = transformPoint4x4(p_orig, projmatrix);
	float p_w = 1.0f / (p_hom.w + 0.0000001f);
	float3 p_proj = { p_hom.x * p_w, p_hom.y * p_w, p_hom.z * p_w };
	p_view = transformPoint4x3(p_orig, viewmatrix);

	if (p_view.z <= 0.2f)// || ((p_proj.x < -1.3 || p_proj.x > 1.3 || p_proj.y < -1.3 || p_proj.y > 1.3)))
	{
		return false;
	}
	return true;
}


__device__ float3 computeCov2DForwardCUDAKernel(const float3& mean, float focal_x, float focal_y, float tan_fovx, float tan_fovy, const float* cov3D, const float* viewmatrix)
{
	// The following models the steps outlined by equations 29
	// and 31 in "EWA Splatting" (Zwicker et al., 2002). 
	// Additionally considers aspect / scaling of viewport.
	// Transposes used to account for row-/column-major conventions.
	float3 t = transformPoint4x3(mean, viewmatrix);

	const float limx = 1.3f * tan_fovx;
	const float limy = 1.3f * tan_fovy;
	const float txtz = t.x / t.z;
	const float tytz = t.y / t.z;
	t.x = min(limx, max(-limx, txtz)) * t.z;
	t.y = min(limy, max(-limy, tytz)) * t.z;

	glm::mat3 J = glm::mat3(
		focal_x / t.z, 0.0f, -(focal_x * t.x) / (t.z * t.z),
		0.0f, focal_y / t.z, -(focal_y * t.y) / (t.z * t.z),
		0, 0, 0);

	glm::mat3 W = glm::mat3(
		viewmatrix[0], viewmatrix[4], viewmatrix[8],
		viewmatrix[1], viewmatrix[5], viewmatrix[9],
		viewmatrix[2], viewmatrix[6], viewmatrix[10]);

	glm::mat3 T = W * J;

	glm::mat3 Vrk = glm::mat3(
		cov3D[0], cov3D[1], cov3D[2],
		cov3D[1], cov3D[3], cov3D[4],
		cov3D[2], cov3D[4], cov3D[5]);

	glm::mat3 cov = glm::transpose(T) * glm::transpose(Vrk) * T;

	// Apply low-pass filter: every Gaussian should be at least
	// one pixel wide/high. Discard 3rd row and column.
	cov[0][0] += 0.3f;
	cov[1][1] += 0.3f;
	return { float(cov[0][0]), float(cov[0][1]), float(cov[1][1]) };
}

// Forward method for converting scale and rotation properties of each
// Gaussian to a 3D covariance matrix in world space. Also takes care
// of quaternion normalization.
__device__ void computeCov3DForwardCUDAKernel(const glm::vec3 scale, const glm::vec4 rot, float* cov3D)
{
	// Create scaling matrix
	glm::mat3 S = glm::mat3(1.0f);
	S[0][0] = scale.x;
	S[1][1] = scale.y;
	S[2][2] = scale.z;

	// Normalize quaternion to get valid rotation
	glm::vec4 q = rot;// / glm::length(rot);
	float r = q.x;
	float x = q.y;
	float y = q.z;
	float z = q.w;

	// Compute rotation matrix from quaternion
	glm::mat3 R = glm::mat3(
		1.f - 2.f * (y * y + z * z), 2.f * (x * y - r * z), 2.f * (x * z + r * y),
		2.f * (x * y + r * z), 1.f - 2.f * (x * x + z * z), 2.f * (y * z - r * x),
		2.f * (x * z - r * y), 2.f * (y * z + r * x), 1.f - 2.f * (x * x + y * y)
	);

	glm::mat3 M = S * R;

	// Compute 3D world covariance matrix Sigma
	glm::mat3 Sigma = glm::transpose(M) * M;

	// Covariance is symmetric, only store upper right
	cov3D[0] = Sigma[0][0];
	cov3D[1] = Sigma[0][1];
	cov3D[2] = Sigma[0][2];
	cov3D[3] = Sigma[1][1];
	cov3D[4] = Sigma[1][2];
	cov3D[5] = Sigma[2][2];
}


template<int C>
__global__ void PreprocessForwardCUDAKernel(int P,
	const dim3 grid,
	const float* orig_points,
	const glm::vec3* scales,
	const glm::vec4* rotations,
	const float* cov3Ds_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const int W, const int H,
	const float focal_x, const float focal_y,
	const float tan_fovx, const float tan_fovy,
	float* depths,
	int32_t* radii,
	float2* means2D,
	float* cov3Ds,
	float3* conics,
	int32_t* tiles_touched)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;
	
	// Initialize radius and touched tiles to 0. If this isn't changed,
	// this Gaussian will not be processed further.
	radii[idx] = 0;
	tiles_touched[idx] = 0;

	// Perform near culling, quit if outside.
	float3 p_view;
	if (!in_frustum(idx, orig_points, viewmatrix, projmatrix, p_view))
		return;

	// Transform point by projecting
	float3 p_orig = { orig_points[3 * idx], orig_points[3 * idx + 1], orig_points[3 * idx + 2] };
	float4 p_hom = transformPoint4x4(p_orig, projmatrix);
	float p_w = 1.0f / (p_hom.w + 0.0000001f);
	float3 p_proj = { p_hom.x * p_w, p_hom.y * p_w, p_hom.z * p_w };
	
	// If 3D covariance matrix is precomputed, use it, otherwise compute
	// from scaling and rotation parameters. 
	const float* cov3D;
	if (cov3Ds_precomp != nullptr)
	{
		cov3D = cov3Ds_precomp + idx * 6;
		cov3Ds[idx * 6 + 0] = cov3D[0];
		cov3Ds[idx * 6 + 1] = cov3D[1];
		cov3Ds[idx * 6 + 2] = cov3D[2];
		cov3Ds[idx * 6 + 3] = cov3D[3];
		cov3Ds[idx * 6 + 4] = cov3D[4];
		cov3Ds[idx * 6 + 5] = cov3D[5];
	}
	else
	{
		computeCov3DForwardCUDAKernel(scales[idx], rotations[idx], cov3Ds + idx * 6);
		cov3D = cov3Ds + idx * 6;
	}
	
	// Compute 2D screen-space covariance matrix
	float3 cov = computeCov2DForwardCUDAKernel(p_orig, focal_x, focal_y, tan_fovx, tan_fovy, cov3D, viewmatrix);
	
	// Invert covariance (EWA algorithm)
	float det = (cov.x * cov.z - cov.y * cov.y);
	if (det == 0.0f)
		return;
	float det_inv = 1.f / det;

	// Compute extent in screen space (by finding eigenvalues of
	// 2D covariance matrix). Use extent to compute a bounding rectangle
	// of screen-space tiles that this Gaussian overlaps with. Quit if
	// rectangle covers 0 tiles. 
	float mid = 0.5f * (cov.x + cov.z);
	float lambda1 = mid + sqrt(max(0.1f, mid * mid - det));
	float lambda2 = mid - sqrt(max(0.1f, mid * mid - det));
	float my_radius = ceil(3.f * sqrt(max(lambda1, lambda2)));
	float2 mean2D = { ndc2Pix(p_proj.x, W), ndc2Pix(p_proj.y, H) };
	uint2 rect_min, rect_max;
	getRect(mean2D, my_radius, rect_min, rect_max, grid);
	if ((rect_max.x - rect_min.x) * (rect_max.y - rect_min.y) == 0)
		return;

	// Store some useful helper data for the next steps.
	depths[idx] = p_view.z;
	radii[idx] = my_radius;
	means2D[idx] = mean2D;
	// Inverse 2D covariance and opacity neatly pack into one float4
	conics[idx] = { cov.z * det_inv, -cov.y * det_inv, cov.x * det_inv };
	tiles_touched[idx] = (rect_max.y - rect_min.y) * (rect_max.x - rect_min.x);
}

__global__ void computeCov2DBackwardCUDAKernel(int P,
	const float3* means3D,
	const int* radii,
	const float* cov3Ds,
	const float focal_x, float focal_y,
	const float tan_fovx, float tan_fovy,
	const float* view_matrix,
	const float3* dL_dconics,
	float3* dL_dmeans3D,
	float* dL_dcov3Ds)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P || !(radii[idx] > 0))
		return;

	// Reading location of 3D covariance for this Gaussian
	const float* cov3D = cov3Ds + 6 * idx;

	// Fetch gradients, recompute 2D covariance and relevant 
	// intermediate forward results needed in the backward.
	float3 mean3D = means3D[idx];
	float3 dL_dconic = dL_dconics[idx];
	float3 t = transformPoint4x3(mean3D, view_matrix);
	
	const float limx = 1.3f * tan_fovx;
	const float limy = 1.3f * tan_fovy;
	const float txtz = t.x / t.z;
	const float tytz = t.y / t.z;
	t.x = min(limx, max(-limx, txtz)) * t.z;
	t.y = min(limy, max(-limy, tytz)) * t.z;
	
	const float x_grad_mul = txtz < -limx || txtz > limx ? 0 : 1;
	const float y_grad_mul = tytz < -limy || tytz > limy ? 0 : 1;

	glm::mat3 J = glm::mat3(focal_x / t.z, 0.0f, -(focal_x * t.x) / (t.z * t.z),
		0.0f, focal_y / t.z, -(focal_y * t.y) / (t.z * t.z),
		0, 0, 0);

	glm::mat3 W = glm::mat3(
		view_matrix[0], view_matrix[4], view_matrix[8],
		view_matrix[1], view_matrix[5], view_matrix[9],
		view_matrix[2], view_matrix[6], view_matrix[10]);

	glm::mat3 Vrk = glm::mat3(
		cov3D[0], cov3D[1], cov3D[2],
		cov3D[1], cov3D[3], cov3D[4],
		cov3D[2], cov3D[4], cov3D[5]);

	glm::mat3 T = W * J;

	glm::mat3 cov2D = glm::transpose(T) * glm::transpose(Vrk) * T;

	// Use helper variables for 2D covariance entries. More compact.
	float a = cov2D[0][0] += 0.3f;
	float b = cov2D[0][1];
	float c = cov2D[1][1] += 0.3f;

	float denom = a * c - b * b;
	float dL_da = 0, dL_db = 0, dL_dc = 0;
	float denom2inv = 1.0f / ((denom * denom) + 0.0000001f);

	if (denom2inv != 0)
	{
		float* dL_dcov3D = dL_dcov3Ds + 6 * idx;
		// Gradients of loss w.r.t. entries of 2D covariance matrix,
		// given gradients of loss w.r.t. conic matrix (inverse covariance matrix).
		// e.g., dL / da = dL / d_conic_a * d_conic_a / d_a
		dL_da = denom2inv * (-c * c * dL_dconic.x + 2 * b * c * dL_dconic.y + (denom - a * c) * dL_dconic.z);
		dL_dc = denom2inv * (-a * a * dL_dconic.z + 2 * a * b * dL_dconic.y + (denom - a * c) * dL_dconic.x);
		dL_db = denom2inv * 2 * (b * c * dL_dconic.x - (denom + 2 * b * b) * dL_dconic.y + a * b * dL_dconic.z);

		// Gradients of loss L w.r.t. each 3D covariance matrix (Vrk) entry, 
		// given gradients w.r.t. 2D covariance matrix (diagonal).
		// cov2D = transpose(T) * transpose(Vrk) * T;
		dL_dcov3D[0] += (T[0][0] * T[0][0] * dL_da + T[0][0] * T[1][0] * dL_db + T[1][0] * T[1][0] * dL_dc);
		dL_dcov3D[3] += (T[0][1] * T[0][1] * dL_da + T[0][1] * T[1][1] * dL_db + T[1][1] * T[1][1] * dL_dc);
		dL_dcov3D[5] += (T[0][2] * T[0][2] * dL_da + T[0][2] * T[1][2] * dL_db + T[1][2] * T[1][2] * dL_dc);

		// Gradients of loss L w.r.t. each 3D covariance matrix (Vrk) entry, 
		// given gradients w.r.t. 2D covariance matrix (off-diagonal).
		// Off-diagonal elements appear twice --> double the gradient.
		// cov2D = transpose(T) * transpose(Vrk) * T;
		dL_dcov3D[1] += 2 * T[0][0] * T[0][1] * dL_da + (T[0][0] * T[1][1] + T[0][1] * T[1][0]) * dL_db + 2 * T[1][0] * T[1][1] * dL_dc;
		dL_dcov3D[2] += 2 * T[0][0] * T[0][2] * dL_da + (T[0][0] * T[1][2] + T[0][2] * T[1][0]) * dL_db + 2 * T[1][0] * T[1][2] * dL_dc;
		dL_dcov3D[4] += 2 * T[0][2] * T[0][1] * dL_da + (T[0][1] * T[1][2] + T[0][2] * T[1][1]) * dL_db + 2 * T[1][1] * T[1][2] * dL_dc;
	}

	// Gradients of loss w.r.t. upper 2x3 portion of intermediate matrix T
	// cov2D = transpose(T) * transpose(Vrk) * T;
	float dL_dT00 = 2 * (T[0][0] * Vrk[0][0] + T[0][1] * Vrk[0][1] + T[0][2] * Vrk[0][2]) * dL_da +
		(T[1][0] * Vrk[0][0] + T[1][1] * Vrk[0][1] + T[1][2] * Vrk[0][2]) * dL_db;
	float dL_dT01 = 2 * (T[0][0] * Vrk[1][0] + T[0][1] * Vrk[1][1] + T[0][2] * Vrk[1][2]) * dL_da +
		(T[1][0] * Vrk[1][0] + T[1][1] * Vrk[1][1] + T[1][2] * Vrk[1][2]) * dL_db;
	float dL_dT02 = 2 * (T[0][0] * Vrk[2][0] + T[0][1] * Vrk[2][1] + T[0][2] * Vrk[2][2]) * dL_da +
		(T[1][0] * Vrk[2][0] + T[1][1] * Vrk[2][1] + T[1][2] * Vrk[2][2]) * dL_db;
	float dL_dT10 = 2 * (T[1][0] * Vrk[0][0] + T[1][1] * Vrk[0][1] + T[1][2] * Vrk[0][2]) * dL_dc +
		(T[0][0] * Vrk[0][0] + T[0][1] * Vrk[0][1] + T[0][2] * Vrk[0][2]) * dL_db;
	float dL_dT11 = 2 * (T[1][0] * Vrk[1][0] + T[1][1] * Vrk[1][1] + T[1][2] * Vrk[1][2]) * dL_dc +
		(T[0][0] * Vrk[1][0] + T[0][1] * Vrk[1][1] + T[0][2] * Vrk[1][2]) * dL_db;
	float dL_dT12 = 2 * (T[1][0] * Vrk[2][0] + T[1][1] * Vrk[2][1] + T[1][2] * Vrk[2][2]) * dL_dc +
		(T[0][0] * Vrk[2][0] + T[0][1] * Vrk[2][1] + T[0][2] * Vrk[2][2]) * dL_db;

	// Gradients of loss w.r.t. upper 3x2 non-zero entries of Jacobian matrix
	// T = W * J
	float dL_dJ00 = W[0][0] * dL_dT00 + W[0][1] * dL_dT01 + W[0][2] * dL_dT02;
	float dL_dJ02 = W[2][0] * dL_dT00 + W[2][1] * dL_dT01 + W[2][2] * dL_dT02;
	float dL_dJ11 = W[1][0] * dL_dT10 + W[1][1] * dL_dT11 + W[1][2] * dL_dT12;
	float dL_dJ12 = W[2][0] * dL_dT10 + W[2][1] * dL_dT11 + W[2][2] * dL_dT12;

	float tz = 1.f / t.z;
	float tz2 = tz * tz;
	float tz3 = tz2 * tz;

	// Gradients of loss w.r.t. transformed Gaussian mean t
	float dL_dtx = x_grad_mul * -focal_x * tz2 * dL_dJ02;
	float dL_dty = y_grad_mul * -focal_y * tz2 * dL_dJ12;
	float dL_dtz = -focal_x * tz2 * dL_dJ00 - focal_y * tz2 * dL_dJ11 + (2 * focal_x * t.x) * tz3 * dL_dJ02 + (2 * focal_y * t.y) * tz3 * dL_dJ12;

	// Account for transformation of mean to t
	// t = transformPoint4x3(mean, view_matrix);
	float3 dL_dmean3D = transformVec4x3Transpose({ dL_dtx, dL_dty, dL_dtz }, view_matrix);

	// Gradients of loss w.r.t. Gaussian means, but only the portion 
	// that is caused because the mean affects the covariance matrix.
	// Additional mean gradient is accumulated in BACKWARD::preprocess.
	dL_dmeans3D[idx] = dL_dmean3D;
}

__device__ void computeCov3DBackwardCUDAKernel(int idx, const glm::vec3 scale, const glm::vec4 rotation, const float* dL_dcov3Ds, glm::vec3* dL_dscales, float4* dL_drotations)
{
	// Recompute (intermediate) results for the 3D covariance computation.
	glm::vec4 q = rotation;// / glm::length(rotation);
	float r = q.x;
	float x = q.y;
	float y = q.z;
	float z = q.w;

	glm::mat3 R = glm::mat3(
		1.f - 2.f * (y * y + z * z), 2.f * (x * y - r * z), 2.f * (x * z + r * y),
		2.f * (x * y + r * z), 1.f - 2.f * (x * x + z * z), 2.f * (y * z - r * x),
		2.f * (x * z - r * y), 2.f * (y * z + r * x), 1.f - 2.f * (x * x + y * y)
	);

	glm::mat3 S = glm::mat3(1.0f);

	glm::vec3 s = scale;
	S[0][0] = s.x;
	S[1][1] = s.y;
	S[2][2] = s.z;

	glm::mat3 M = S * R;

	const float* dL_dcov3D = dL_dcov3Ds + 6 * idx;

	glm::vec3 dunc(dL_dcov3D[0], dL_dcov3D[3], dL_dcov3D[5]);
	glm::vec3 ounc = 0.5f * glm::vec3(dL_dcov3D[1], dL_dcov3D[2], dL_dcov3D[4]);

	// Convert per-element covariance loss gradients to matrix form
	glm::mat3 dL_dSigma = glm::mat3(
		dL_dcov3D[0], 0.5f * dL_dcov3D[1], 0.5f * dL_dcov3D[2],
		0.5f * dL_dcov3D[1], dL_dcov3D[3], 0.5f * dL_dcov3D[4],
		0.5f * dL_dcov3D[2], 0.5f * dL_dcov3D[4], dL_dcov3D[5]
	);

	// Compute loss gradient w.r.t. matrix M
	// dSigma_dM = 2 * M
	glm::mat3 dL_dM = 2.0f * M * dL_dSigma;

	glm::mat3 Rt = glm::transpose(R);
	glm::mat3 dL_dMt = glm::transpose(dL_dM);

	// Gradients of loss w.r.t. scale
	glm::vec3* dL_dscale = dL_dscales + idx;
	dL_dscale->x = glm::dot(Rt[0], dL_dMt[0]);
	dL_dscale->y = glm::dot(Rt[1], dL_dMt[1]);
	dL_dscale->z = glm::dot(Rt[2], dL_dMt[2]);

	dL_dMt[0] *= s.x;
	dL_dMt[1] *= s.y;
	dL_dMt[2] *= s.z;

	// Gradients of loss w.r.t. normalized quaternion
	glm::vec4 dL_dq;
	dL_dq.x = 2 * z * (dL_dMt[0][1] - dL_dMt[1][0]) + 2 * y * (dL_dMt[2][0] - dL_dMt[0][2]) + 2 * x * (dL_dMt[1][2] - dL_dMt[2][1]);
	dL_dq.y = 2 * y * (dL_dMt[1][0] + dL_dMt[0][1]) + 2 * z * (dL_dMt[2][0] + dL_dMt[0][2]) + 2 * r * (dL_dMt[1][2] - dL_dMt[2][1]) - 4 * x * (dL_dMt[2][2] + dL_dMt[1][1]);
	dL_dq.z = 2 * x * (dL_dMt[1][0] + dL_dMt[0][1]) + 2 * r * (dL_dMt[2][0] - dL_dMt[0][2]) + 2 * z * (dL_dMt[1][2] + dL_dMt[2][1]) - 4 * y * (dL_dMt[2][2] + dL_dMt[0][0]);
	dL_dq.w = 2 * r * (dL_dMt[0][1] - dL_dMt[1][0]) + 2 * x * (dL_dMt[2][0] + dL_dMt[0][2]) + 2 * y * (dL_dMt[1][2] + dL_dMt[2][1]) - 4 * z * (dL_dMt[1][1] + dL_dMt[0][0]);

	// Gradients of loss w.r.t. unnormalized quaternion
	dL_drotations[idx] = { dL_dq.x, dL_dq.y, dL_dq.z, dL_dq.w };  //dnormvdv(float4{ rot.x, rot.y, rot.z, rot.w }, float4{ dL_dq.x, dL_dq.y, dL_dq.z, dL_dq.w });
}

template<int C>
__global__ void PreprocessBackwardCUDAKernel(
	int P, 
	const float3* means3D,
	const int* radii,
	const glm::vec3* scales,
	const glm::vec4* rotations,
	const float* projmatrix,
	const float2* dL_dmeans2D,
	glm::vec3* dL_dmeans3D,
	float* dL_dcov3Ds,
	glm::vec3* dL_dscales,
	float4* dL_drotations)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P || !(radii[idx] > 0))
		return;

	float3 mean3D = means3D[idx];

	// Taking care of gradients from the screenspace points
	float4 m_hom = transformPoint4x4(mean3D, projmatrix);
	float m_w = 1.0f / (m_hom.w + 0.0000001f);

	// Compute loss gradient w.r.t. 3D means due to gradients of 2D means
	// from rendering procedure
	glm::vec3 dL_dmean3D;
	float mul1 = (projmatrix[0] * mean3D.x + projmatrix[4] * mean3D.y + projmatrix[8] * mean3D.z + projmatrix[12]) * m_w * m_w;
	float mul2 = (projmatrix[1] * mean3D.x + projmatrix[5] * mean3D.y + projmatrix[9] * mean3D.z + projmatrix[13]) * m_w * m_w;
	dL_dmean3D.x = (projmatrix[0] * m_w - projmatrix[3] * mul1) * dL_dmeans2D[idx].x + (projmatrix[1] * m_w - projmatrix[3] * mul2) * dL_dmeans2D[idx].y;
	dL_dmean3D.y = (projmatrix[4] * m_w - projmatrix[7] * mul1) * dL_dmeans2D[idx].x + (projmatrix[5] * m_w - projmatrix[7] * mul2) * dL_dmeans2D[idx].y;
	dL_dmean3D.z = (projmatrix[8] * m_w - projmatrix[11] * mul1) * dL_dmeans2D[idx].x + (projmatrix[9] * m_w - projmatrix[11] * mul2) * dL_dmeans2D[idx].y;

	// That's the second part of the mean gradient. Previous computation
	// of cov2D and following SH conversion also affects it.
	dL_dmeans3D[idx] += dL_dmean3D;

	// Compute gradient updates due to computing covariance from scale/rotation
	if (scales)
		computeCov3DBackwardCUDAKernel(idx, scales[idx], rotations[idx], dL_dcov3Ds, dL_dscales, dL_drotations);
}

void PreprocessForwardCUDA(
	const int P, 
	const int width, const int height,
	const float* means3D,
	const float* scales,
	const float* rotations,
	const float* cov3Ds_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const float tan_fovx, const float tan_fovy,
	float* depths,
	int32_t* radii,
	float2* means2D,
	float* cov3Ds,
	float3* conics,
	int32_t* tiles_touched,
	const bool debug){
    const float focal_y = height / (2.0f * tan_fovy);
	const float focal_x = width / (2.0f * tan_fovx);

	const dim3 tile_grid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y, 1);

	// Run preprocessing per-Gaussian (transformation, bounding, conversion of SHs to RGB)
	PreprocessForwardCUDAKernel<NUM_CHANNELS> <<<(P + 255) / 256, 256 >>> (
		P,
		tile_grid,
		means3D,
		(glm::vec3*)scales,
		(glm::vec4*)rotations,
		cov3Ds_precomp,
		viewmatrix, projmatrix,
		width, height,
		focal_x, focal_y,
		tan_fovx, tan_fovy,
		depths,
		radii,
		means2D,
		cov3Ds,
		conics,
		tiles_touched
	);
}

void PreprocessBackwardCUDA(
	const int P, 
	const int width, int height,
	const float3* means3D,
	const float* scales,
	const float* rotations,
	const float* cov3Ds,
	const float* cov3Ds_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const float tan_fovx, float tan_fovy,
	const int* radii,
	const float* dL_ddepths,
	const float2* dL_dmeans2D,
	const float3* dL_dconics,
	float* dL_dmeans3D,
	float* dL_dscales,
	float4* dL_drotations,
	float* dL_dcov3Ds,  // need to add grad on this tensor, because cov3Ds is also a output in forward (while not consider the effect from cov2D)
	bool debug)
{
	const float focal_y = height / (2.0f * tan_fovy);
	const float focal_x = width / (2.0f * tan_fovx);

	const dim3 tile_grid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y, 1);

	// Take care of the rest of preprocessing. Was the precomputed covariance
	// given to us or a scales/rot pair? If precomputed, pass that. If not,
	// use the one we computed ourselves.
	computeCov2DBackwardCUDAKernel <<<(P + 255) / 256, 256 >>> (
		P,
		means3D,
		radii,
		cov3Ds,
		focal_x,
		focal_y,
		tan_fovx,
		tan_fovy,
		viewmatrix,
		dL_dconics,
		(float3*)dL_dmeans3D,
		dL_dcov3Ds);

	// Propagate gradients for remaining steps: finish 3D mean gradients,
	// propagate color gradients to SH (if desireD), propagate 3D covariance
	// matrix gradients to scale and rotation.
	PreprocessBackwardCUDAKernel<NUM_CHANNELS> <<< (P + 255) / 256, 256 >>> (
		P,
		means3D,
		radii,
		(const glm::vec3*)scales,
		(const glm::vec4*)rotations,
		projmatrix,
		dL_dmeans2D,
		(glm::vec3*)dL_dmeans3D,
		dL_dcov3Ds,
		(glm::vec3*)dL_dscales,
		dL_drotations);
}